#include "hip/hip_runtime.h"



#include <stdio.h>
#include <stdlib.h> 
#include <math.h>


#include "CudaSupport.h"
#include "KMeansInnerMethods.h"



#define LOAD_BALANCING_FACTOR 2
#define THREADS_PER_BLOCK 1000

int findnumOfExacutions(int numCords);
float findMaxDistPowerInArr(float* diatArr, int numElements);

// populate the max dist arry only with the power ^2 of the distances, and the actual distances
__global__ void findMaxDistKernel(Coordinates*cords, const int numCords,float* maxDistArr,int exacutionIndex)
{

	int id=  threadIdx.x+ exacutionIndex*THREADS_PER_BLOCK;
 int cordsToCalc = numCords /LOAD_BALANCING_FACTOR ;

 if (id +1> numCords)
   return;

 int count=0 , CordIndex=id+1;
	float tempMax ;


	for (CordIndex; CordIndex < numCords&&count<cordsToCalc; CordIndex++)
	{
		tempMax = (cords[id].x - cords[CordIndex].x)*(cords[id].x - cords[CordIndex].x) + (cords[id].y - cords[CordIndex].y)*(cords[id].y - cords[CordIndex].y);

		if (tempMax>maxDistArr[id]) 
			maxDistArr[id] = tempMax;

		count++;
    
	}

 //cyclic manner 

	for (CordIndex = 0; count < cordsToCalc; CordIndex++)
	{
		tempMax = (cords[id].x - cords[CordIndex].x)*(cords[id].x - cords[CordIndex].x) + (cords[id].y - cords[CordIndex].y)*(cords[id].y - cords[CordIndex].y);

		if (tempMax>maxDistArr[id])
			maxDistArr[id] = tempMax;

		count++;
	       
	
	}
	
}

hipError_t calcDiameterWithCuda(Coordinates* cords,int numCords,float* resultsPtr)
{

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	
	Coordinates* cordOnDev;
	float* maxsOnDev,*maxsOnHost;


	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return cudaStatus;
	}


	cudaStatus = hipMalloc((void**)&cordOnDev, numCords * sizeof(Coordinates));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMalloc failed!");
		hipFree(cordOnDev);
		return cudaStatus;
	}



	cudaStatus = hipMalloc((void**)&maxsOnDev, numCords * sizeof(float));
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipMalloc failed!");
		hipFree(cordOnDev);
		hipFree(maxsOnDev);
		return cudaStatus;
	}

	hipMemset(maxsOnDev, 0, numCords * sizeof(float));

	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemset failed!");
		hipFree(cordOnDev);
		hipFree(maxsOnDev);
		return cudaStatus;
	}



	cudaStatus = hipMemcpy(cordOnDev, cords,numCords * sizeof(Coordinates), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(cordOnDev);
		hipFree(maxsOnDev);
		return cudaStatus;
	}


	int exacutionIndex;
	int numOfExacutions = findnumOfExacutions(numCords);
 float t1, t2;


	for (exacutionIndex =0; exacutionIndex < numOfExacutions; exacutionIndex++)
	{
 
		findMaxDistKernel << <1, THREADS_PER_BLOCK >> > (cordOnDev, numCords , maxsOnDev, exacutionIndex);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "calcDiameterWithCuda launch failed: %s\n", hipGetErrorString(cudaStatus));
			hipFree(cordOnDev);
			hipFree(maxsOnDev);
			return  cudaStatus;
		}


		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			hipFree(cordOnDev);
			hipFree(maxsOnDev);
			return cudaStatus;
		}

	
 
 }

	maxsOnHost = (float*)malloc(sizeof(float)*numCords);
	
	cudaStatus = hipMemcpy(maxsOnHost, maxsOnDev, numCords * sizeof(float), hipMemcpyDeviceToHost);

             
	*resultsPtr=(float)sqrt((double)findMaxDistPowerInArr(maxsOnHost, numCords));
	
	hipFree(maxsOnDev);
	hipFree(cordOnDev);
	free(maxsOnHost);

	return cudaStatus;



}



float findMaxDistPowerInArr (float* diatArr,int numElements)
{
	int i;
	float maxDist = 0;

	for (i = 0; i < numElements; i++) 
	{
	
		if (maxDist < diatArr[i])
			maxDist = diatArr[i];
	
	}


	return maxDist;

}


int findnumOfExacutions(int numCords)
{

	if (numCords%THREADS_PER_BLOCK == 0) 
	{

		return numCords / THREADS_PER_BLOCK;
	}
	else { return (numCords / THREADS_PER_BLOCK) + 1; }
	
}


void freeDeviceMeme(void ** ptrArr,int numPtrs)
{

	int i = 0;

	for (i = 0; i < numPtrs; i++)
		hipFree(ptrArr[i]);

}




//(int *c, const int *a, const int *b, unsigned int size)
/*  int *dev_a = 0;
int *dev_b = 0;
int *dev_c = 0;
hipError_t cudaStatus;

// Choose which GPU to run on, change this on a multi-GPU system.
cudaStatus = hipSetDevice(0);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
goto Error;
}

// Allocate GPU buffers for three vectors (two input, one output)    .
cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMalloc failed!");
goto Error;
}

cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMalloc failed!");
goto Error;
}

cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMalloc failed!");
goto Error;
}

// Copy input vectors from host memory to GPU buffers.
cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMemcpy failed!");
goto Error;
}

cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMemcpy failed!");
goto Error;
}

// Launch a kernel on the GPU with one thread for each element.
addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

// Check for any errors launching the kernel
cudaStatus = hipGetLastError();
if (cudaStatus != hipSuccess) {
fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
goto Error;
}

// hipDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
cudaStatus = hipDeviceSynchronize();
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
goto Error;
}

// Copy output vector from GPU buffer to host memory.
cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMemcpy failed!");
goto Error;
}

Error:
hipFree(dev_c);
hipFree(dev_a);
hipFree(dev_b);

return cudaStatus;
*/

